/**********************************************************************************/
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020, 2021 JetBrains-Research                                    */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/**********************************************************************************/

#include <cuda/matrix_csr.hpp>
#include <core/error.hpp>
#include <utils/timer.hpp>
#include <algorithm>

namespace spbla {

    MatrixCsr::MatrixCsr(size_t nrows, size_t ncols, Instance &instance) : mInstance(instance) {
        mNrows = nrows;
        mNcols = ncols;
    }

    void MatrixCsr::setElement(index i, index j) {
        RAISE_ERROR(NotImplemented, "This function is not supported for this matrix class");
    }

    void MatrixCsr::clone(const MatrixBase &otherBase) {
        auto other = dynamic_cast<const MatrixCsr*>(&otherBase);

        CHECK_RAISE_ERROR(other != nullptr, InvalidArgument, "Passed matrix does not belong to csr matrix class");
        CHECK_RAISE_ERROR(other != this, InvalidArgument, "Matrices must differ");

        size_t M = other->getNrows();
        size_t N = other->getNcols();

        assert(this->getNrows() == M);
        assert(this->getNcols() == N);

        if (other->isMatrixEmpty()) {
            mMatrixImpl.zero_dim();
            return;
        }

        this->mMatrixImpl = other->mMatrixImpl;
    }

    void MatrixCsr::resizeStorageToDim() const {
        if (mMatrixImpl.is_zero_dim()) {
            // If actual storage was not allocated, allocate one for an empty matrix
            mMatrixImpl = std::move(MatrixImplType(mNrows, mNcols));
        }
    }

    void MatrixCsr::clearAndResizeStorageToDim() const {
        if (mMatrixImpl.m_vals > 0) {
            // Release only if have some nnz values
            mMatrixImpl.zero_dim();
        }

        // Normally resize if no storage is actually allocated
        this->resizeStorageToDim();
    }

    index MatrixCsr::getNrows() const {
        return mNrows;
    }

    index MatrixCsr::getNcols() const {
        return mNcols;
    }

    index MatrixCsr::getNvals() const {
        return mMatrixImpl.m_vals;
    }

    bool MatrixCsr::isStorageEmpty() const {
        return mMatrixImpl.is_zero_dim();
    }

    bool MatrixCsr::isMatrixEmpty() const {
        return mMatrixImpl.m_vals == 0;
    }

    void MatrixCsr::transferToDevice(const std::vector<index> &rowOffsets, const std::vector<index> &colIndices) const {
        // Create device buffers and copy data from the cpu side
        thrust::device_vector<index, DeviceAlloc<index>> rowsDeviceVec(rowOffsets.size());
        thrust::device_vector<index, DeviceAlloc<index>> colsDeviceVec(colIndices.size());

        thrust::copy(rowOffsets.begin(), rowOffsets.end(), rowsDeviceVec.begin());
        thrust::copy(colIndices.begin(), colIndices.end(), colsDeviceVec.begin());

        // Move actual data to the matrix implementation
        mMatrixImpl = std::move(MatrixImplType(std::move(colsDeviceVec), std::move(rowsDeviceVec), getNrows(), getNcols(), colIndices.size()));
    }

    void MatrixCsr::transferFromDevice(std::vector<index> &rowOffsets, std::vector<index> &colIndices) const {
        rowOffsets.resize(mMatrixImpl.m_row_index.size());
        colIndices.resize(mMatrixImpl.m_col_index.size());

        thrust::copy(mMatrixImpl.m_row_index.begin(), mMatrixImpl.m_row_index.end(), rowOffsets.begin());
        thrust::copy(mMatrixImpl.m_col_index.begin(), mMatrixImpl.m_col_index.end(), colIndices.begin());
    }

}