/**********************************************************************************/
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020, 2021 JetBrains-Research                                    */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/**********************************************************************************/

#include <cuda/instance.hpp>
#include <cuda/matrix_dense.hpp>
#include <core/error.hpp>
#include <string>
#include <cstring>

namespace spbla {

    Instance::~Instance() {
        assert(mHostAllocCount == 0);
        assert(mDeviceAllocCount == 0);

        gInstance = nullptr;
    }

    void Instance::allocateOnGpu(void* &ptr, size_t size) const {
        hipError_t error;

        switch (mMemoryType) {
            case MemType::Default:
                error = hipMalloc(&ptr, size);
                break;
            case MemType::Managed:
                error = hipMallocManaged(&ptr, size);
                break;
            default:
                RAISE_ERROR(MemOpFailed, "Failed to fined suitable allocator");
        }

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to allocate Gpu memory: "} + hipGetErrorString(error);
            RAISE_ERROR(MemOpFailed, message);
        }

        mDeviceAllocCount++;
    }

    void Instance::deallocateOnGpu(void* ptr) const {
        hipError_t error = hipFree(ptr);

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to deallocate Gpu memory: "} + hipGetErrorString(error);
            RAISE_ERROR(MemOpFailed, message);
        }

        mDeviceAllocCount--;
    }

    void Instance::syncHostDevice() const {
        hipError_t error = hipDeviceSynchronize();

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to synchronize host and device: "} + hipGetErrorString(error);
            RAISE_ERROR(DeviceError, message);
        }
    }

    bool Instance::isCudaDeviceSupported() {
        int device;
        hipError_t error = hipGetDevice(&device);
        return error == hipSuccess;
    }

    void Instance::queryDeviceCapabilities(spbla_DeviceCaps &deviceCaps) {
        const unsigned long long KiB = 1024;

        int device;
        hipError_t error = hipGetDevice(&device);

        if (error == hipSuccess) {
            hipDeviceProp_t deviceProp{};
            error = hipGetDeviceProperties(&deviceProp, device);

            if (error == hipSuccess) {
                strcpy(deviceCaps.name, deviceProp.name);
                deviceCaps.cudaSupported = true;
                deviceCaps.minor = deviceProp.minor;
                deviceCaps.major = deviceProp.major;
                deviceCaps.warp = deviceProp.warpSize;
                deviceCaps.globalMemoryKiBs = deviceProp.totalGlobalMem / KiB;
                deviceCaps.sharedMemoryPerMultiProcKiBs = deviceProp.sharedMemPerMultiprocessor / KiB;
                deviceCaps.sharedMemoryPerBlockKiBs = deviceProp.sharedMemPerBlock / KiB;
            }
        }
    }

}
